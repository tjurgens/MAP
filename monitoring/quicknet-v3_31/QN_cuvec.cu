#include "hip/hip_runtime.h"
const char* QN_cuvec_rcsid =
    "$Header: /u/drspeech/repos/quicknet2/QN_cuvec.cu,v 1.1 2011/03/10 00:27:57 davidj Exp $";

/* Must include the config.h file first */
#include <QN_config.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "QN_types.h"
#include "QN_cuvec.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>


__global__ void
qn_devnv_copy_vf_vf(size_t len, const float* from, float* to)
{
    size_t i;

    for (i=len; i!=0; i--)
	*to++ = *from++;
}

__global__ void
qn_devnv_sub_vfvf_vf(size_t n,
		     const float *in_vec1, const float *in_vec2,
		     float *res_vec)
{
    size_t i;
    for (i=n; i!=0; i--)
        (*res_vec++) = *(in_vec1++) - (*in_vec2++);
}

__global__ void
qn_devnv_mul_vfvf_vf(size_t n,
		     const float *in_vec1, const float *in_vec2,
		     float *res_vec)
{
    size_t i;
    for (i=n; i!=0; i--)
        (*res_vec++) = (*in_vec1++) * (*in_vec2++);
}

__global__ void
qn_devnv_mulacc_vff_vf(size_t n, const float *in, float scale, float *acc)
{
    size_t i;

    for (i=n; i!=0; i--)
        (*acc++) += scale * (*in++);
}

__global__ void
qn_devnv_dsigmoid_vf_vf(size_t n, const float* in_vec, float* out_vec)
{
    size_t i;

    for (i=n; i!=0; i--)
    {
        const float y = *in_vec++;

	*out_vec++ = (1.0f - y) * y;
    }
}


__global__ void
qn_devnv_sigmoid_vf_vf(int n, const float* in_vec, float* out_vec)
{
    size_t i;

    for (i=0; i<n; i++)
    {
	*out_vec++ = qn_devin_sigmoid_f_f(*in_vec++);
    }
}

__global__ void
qn_kern_sub_vfvf_vf(int n, const float* in_vec1, const float* in_vec2,
		      float* out_vec)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i<n)
	out_vec[i] = in_vec1[i] - in_vec2[i];
}

__global__ void
qn_kern_mul_vfvf_vf(int n, const float* in_vec1, const float* in_vec2,
		      float* out_vec)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i<n)
	out_vec[i] = in_vec1[i] * in_vec2[i];
}

__global__ void
qn_kern_sigmoid_vf_vf(int n, const float* in_vec, float* out_vec)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i<n)
	out_vec[i] = 1.0f/(1.0f + __expf(-in_vec[i]));
}

__global__ void
qn_kern_tanh_vf_vf(int n, const float* in_vec, float* out_vec)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i<n)
	out_vec[i] = tanh(in_vec[i]);
}


__global__ void
qn_kern_dsigmoid_vf_vf(int n, const float* in_vec, float* out_vec)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i<n)
    {
	const float y = in_vec[i];
	out_vec[i] = (1.0f - y) * y;
    }
}

__global__ void
qn_kern_dtanh_vf_vf(int n, const float* in_vec, float* out_vec)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i<n)
    {
	const float y = in_vec[i];
	out_vec[i] = (1.0f - y) * (1.0f + y);
    }
}

__global__ void
qn_kern_copy_vf_mf(int mat_height, int vec_len,
		   const float*vec, float* mat)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (col<vec_len)
    {
	int j;
	float val = vec[col];
	float* top = &mat[col];
	for (j=mat_height; j!=0; j--)
	{
	    *top = val;
	    top += vec_len;
	}
    }
}

// Kernel to sum columns in a matrix
// Do each column sum in its own thread
__global__ void
qn_kern_sumcol_mf_vf(int rows, int cols,
		     const float* in, float* res)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (col < cols)
    {
	int j;
	const float* fromp = &in[col];
	float* top = &res[col];
	
	(*top) = (*fromp);
	fromp+=cols;
	for (j=rows-1; j!=0; j--)
	{
	    (*top) += (*fromp);
	    fromp+=cols;
	}
    }
}

// Kernel for multisoftmax - several softmaxes at once
// Do each softmax row in its own thread
__global__ void
qn_kern_multisoftmax_mf_mf(int rows, int cols, const float* in_vec,
			   float* out_vec)
{
    int row = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (row < rows)
    {
	int i;
	const int index = row * cols;
	const float* invec = &in_vec[index];
        float* outvec = &out_vec[index];
	const float* inptr;
	float* outptr;

	// First find the max of each vector
	float max;
	
	inptr = invec;
	max = *inptr++;
	for (i=cols-1; i!=0; i--)
	{
	    float val;

	    val = *inptr++;
	    if (val>max)
		max = val;
	}
	// Now put exp(in-max) in out
	inptr = invec;
	outptr = outvec;
	float sumexp = 0;
	for (i=cols; i!=0; i--)
	{
	    float f, e;
	    
	    f = *inptr++;
	    e = expf(f - max);
	    *outptr++ = e;
	    sumexp += e;
	}
	// Now scale the output
	float scale = 1.0f/sumexp;
	outptr = outvec;
	for (i=cols; i!=0; i--)
	{
	    *outptr = (*outptr) * scale;
	    outptr++;
	}
    }
}

__global__ void
qn_devnv_softmax_vf_vf(int n, const float* in_vec, float* out_vec)
{
    float max;
    float min;
    float sumexp = 0.0f;	/* Sum of exponents */
    float scale;		/* 1/sum of exponents */
    size_t i;

    qn_devin_maxmin_vf_ff(n, in_vec, &max, &min);	/* Find constant bias. */
    for (i=0; i<n; i++)
    {
	float f;		/* Input value. */
	float e;		/* Exponent of current value. */

	f = in_vec[i];
	e = expf(f - max);
	out_vec[i] = e;
	sumexp += e;
    }
    scale = 1.0f/sumexp;
    for (i=0; i<n; i++)
    {
	out_vec[i] = out_vec[i] * scale;
    }
}

__global__ void
qn_devnv_copy_vf_mf(size_t mat_height, size_t vec_len, const float* vec,
		    float* mat)
{
    size_t i, j;

    const float* vec_ptr;
    float* mat_ptr = mat;

    for (i=mat_height; i!=0; i--)
    {
	vec_ptr = vec;
	for (j=vec_len; j!=0; j--)
	{
	    (*mat_ptr++) = (*vec_ptr++);
	}
    }
}

__global__ void
qn_devnv_sumcol_mf_vf(size_t rows, size_t cols, const float* in, float* res)
{
    const float *const res_end_b8p = res + (cols & ~7);
    const float *const res_end_p = res + cols;
    float* res_p;
    const float* in_p = in;
    size_t i;

    /* Initialize the result */
    res_p = res;
    while(res_p != res_end_b8p)
    {
	res_p[0] = in_p[0];
	res_p[1] = in_p[1];
	res_p[2] = in_p[2];
	res_p[3] = in_p[3];
	res_p[4] = in_p[4];
	res_p[5] = in_p[5];
	res_p[6] = in_p[6];
	res_p[7] = in_p[7];
	res_p += 8;
	in_p += 8;
    }
    while (res_p != res_end_p)
    {
	(*res_p++) = (*in_p++);
    }
    /* The main loop */
    for (i=1; i!=rows; i++)
    {
	res_p = res;
	while(res_p != res_end_b8p)
	{
	    res_p[0] += in_p[0];
	    res_p[1] += in_p[1];
	    res_p[2] += in_p[2];
	    res_p[3] += in_p[3];
	    res_p[4] += in_p[4];
	    res_p[5] += in_p[5];
	    res_p[6] += in_p[6];
	    res_p[7] += in_p[7];
	    res_p += 8;
	    in_p += 8;
	}
	while (res_p != res_end_p)
	{
	    (*res_p++) += (*in_p++);
	}
    }
}


__global__ void
qn_devnv_mul_mfmf_mf(size_t Sm,size_t Sk,size_t Sn,
		  const float *A,const float *B,float *C)
{
  size_t i,j,k;
  for (i=0;i<Sm;i++)
  {
      for (j=0;j<Sn;j++)
      {
	  C[i*Sn+j] = 0.0f;
	  for (k=0;k<Sk;k++)
	      C[i*Sn+j] += A[i*Sk+k]*B[k*Sn+j];
      }
  }
}

__global__ void
qn_devnv_mulntacc_mfmf_mf(size_t Sm,size_t Sk,size_t Sn,
			  const float *A,const float *B,float *C)
{
  size_t i,j,k;
  for (i=0;i<Sm;i++)
    for (j=0;j<Sn;j++)
      for (k=0;k<Sk;k++)
        C[i*Sn+j] += A[i*Sk+k]*B[j*Sk+k];
}


__global__ void
qn_devnv_multnacc_fmfmf_mf(size_t Sk,size_t Sm,size_t Sn, float scale,
			   const float *A,const float *B,float *C)
{
    size_t i,j,k;
    float acc;

    for (i=0;i<Sm;i++)
    {
	for (j=0;j<Sn;j++)
	{
	    acc = 0.0f;
	    for (k=0;k<Sk;k++)
		acc += A[k*Sm+i]*B[k*Sn+j];
	    C[i*Sn+j] += acc * scale;
	}
    }
}


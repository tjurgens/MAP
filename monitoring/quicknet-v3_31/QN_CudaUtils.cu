const char* QN_CudaUtils_rcsid =
    "$Header: /u/drspeech/repos/quicknet2/QN_CudaUtils.cu,v 1.4 2011/05/21 00:10:35 davidj Exp $";

/* Must include the config.h file first */
#include <QN_config.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "QN_types.h"
#include "QN_Logger.h"
#include "QN_CudaUtils.h"
#include "QN_fltvec.h"
#include "QN_intvec.h"
#include "QN_cuvec.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>


static int QN_cuda_inited = QN_FALSE;

size_t
QN_cuda_maxvec(void)
{
	return ((size_t) QN_CUDA_NTHREADS * (size_t) QN_CUDA_MAXBLOCKS) -1;
}

void
QN_cuda_init(void)
{

    if (QN_cuda_inited == QN_FALSE)
    {
	QN_cuda_inited = QN_TRUE;

	hipError_t e;
	float* test;
	// A malloc test to trap lack of CUDA resources cleanly
	e = hipMalloc((void **) &test, 4);
	if (e!=hipSuccess)
	    QN_ERROR(NULL, hipGetErrorString(e));
	e = hipFree(test);
	if (e!=hipSuccess)
	    QN_ERROR("QN_cuda_init", hipGetErrorString(e));

	hipblasStatus_t eb;
	eb = cublasInit();
	if (eb!=HIPBLAS_STATUS_SUCCESS)
	    QN_ERROR("QN_cuda_init", "failed to initalize CUDA BLAS");

	// Make sure the current device is good enough
	int dev;
	e = hipGetDevice(&dev);
	if (e!=hipSuccess)
	    QN_ERROR("QN_cuda_init", "failed to get current device");

	struct hipDeviceProp_t props;
	e = hipGetDeviceProperties(&props, dev);
	if (e!=hipSuccess)
	    QN_ERROR("QN_cuda_init", "failed to get device properties");
	if (props.major<1)
	    QN_ERROR("QN_cuda_init", "need minimum compute capability 1.x");
	if (props.major==1 && props.minor<3)
	    QN_WARN("QN_cuda_init", "QuickNet not tested on compute capability <1.3");
	if (QN_CUDA_NTHREADS > props.maxThreadsDim[0])
	    QN_ERROR("QN_cuda_init", "need GPU with maxThreadsDim[0]>=%i", QN_CUDA_NTHREADS);
	if (QN_CUDA_MAXBLOCKS > props.maxGridSize[0])
	    QN_ERROR("QN_cuda_init", "need GPU with maxGridSize[0]>=%i", QN_CUDA_MAXBLOCKS);
    }
}

void
QN_cuda_check(void)
{

    if (QN_cuda_inited == QN_TRUE)
    {
	hipblasStatus_t eb;
	eb = cublasGetError();
	if (eb!=HIPBLAS_STATUS_SUCCESS)
	    QN_ERROR("QN_cuda_check", "accumulated cublas error detected");
    }
}

void
QN_cuda_shutdown(void)
{

    if (QN_cuda_inited == QN_TRUE)
    {
	hipblasStatus_t eb;
	eb = cublasGetError();
	if (eb!=HIPBLAS_STATUS_SUCCESS)
	    QN_ERROR("QN_cuda_shutdown", "accumulated cublas error detected");
	eb = cublasShutdown();
	if (eb!=HIPBLAS_STATUS_SUCCESS)
	    QN_ERROR("QN_cuda_shutdown", "failed to shutdown CUDA BLAS");
	QN_cuda_inited = QN_FALSE;
    }
}

static const char*
cuda_modet_to_txt(int mode)
{
    const char* res;

    switch(mode)
    {
    case hipComputeModeDefault:
	res = "DEFAULT";
	break;
    case hipComputeModeExclusive:
	res = "EXCLUSIVE";
	break;
    case hipComputeModeProhibited:
	res = "PROHIBITED";
	break;
    default:
	res = "WEIRD";
	break;
    }
    return res;
}


const char*
QN_cuda_version(void)
{
    hipError_t e;
    static char cuda_ver_info[1024];
    char* ptr = cuda_ver_info;
    int devcount = 0;

    e = hipGetDeviceCount(&devcount);
    if (e!=hipSuccess)
    {
	ptr += sprintf(ptr, "CUDA error: %s.\n", hipGetErrorString(e));
    }

    if (devcount!=0)
    {
	int runver = 0;
	int drvver = 0;
	e = hipDriverGetVersion(&drvver);
	if (e!=hipSuccess)
	    QN_ERROR("QN_cuda_version", "failed to get CUDA driver version");
	e = hipRuntimeGetVersion(&runver);
	if (e!=hipSuccess)
	    QN_ERROR("QN_cuda_version", "failed to get CUDA runtime version");
	ptr += sprintf(ptr, "CUDA runtime version: %i\nCUDA driver version: %i.\n", runver, drvver);
    }
    ptr += sprintf(ptr, "CUDA available device count: %i.", devcount);

    int i;
    struct hipDeviceProp_t props;

    for (i=0; i<devcount; i++)
    {
	e = hipGetDeviceProperties(&props, i);
	if (e!=hipSuccess)
	    QN_ERROR("QN_cuda_version", "failed to get device properties");
	ptr += sprintf(ptr,"\nCUDA device %i: %s %iMB @ %.2f GHz [%s].", i,
		       props.name,props.totalGlobalMem/(1024*1024),
		       ((double) props.clockRate)/1e6,
		       cuda_modet_to_txt(props.computeMode));
			
    }

    return cuda_ver_info;
}

//
int
QN_cuda_current_device(void)
{
    int dev;
    hipError_t e;

    e = hipGetDevice(&dev);
    if (e!=hipSuccess)
	QN_ERROR("QN_cuda_current_device", "failed to get current device");
    return dev;
}

const char*
QN_cublas_error_string(int e)
{
    const char* s;

    switch(e)
    {
    case HIPBLAS_STATUS_SUCCESS:
	s = "cublas ";
	break;
    case HIPBLAS_STATUS_NOT_INITIALIZED:
	s = "cublas not initialized";
	break;
    case HIPBLAS_STATUS_ALLOC_FAILED:
	s = "cublas alloc failed";
	break;
    case HIPBLAS_STATUS_INVALID_VALUE:
	s = "cublas invalid value";
	break;
    case HIPBLAS_STATUS_ARCH_MISMATCH:
	s = "cublas arch mismatch";
	break;
    case HIPBLAS_STATUS_MAPPING_ERROR:
	s = "cublas mapping error";
	break;
    case HIPBLAS_STATUS_EXECUTION_FAILED:
	s = "cublas execution failed";
	break;
    case HIPBLAS_STATUS_INTERNAL_ERROR:
	s = "cublas internal error";
	break;
    default:
	s = "cublas unknown error";
	break;
    }

    return s;
}


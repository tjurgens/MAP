#include "hip/hip_runtime.h"
const char* QN_MLP_BunchCudaVar_rcsid =
    "$Header: /u/drspeech/repos/quicknet2/QN_MLP_BunchCudaVar.cu,v 1.5 2011/05/24 02:03:14 davidj Exp $";

/* Must include the config.h file first */
#include <QN_config.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "QN_types.h"
#include "QN_Logger.h"
#include "QN_CudaUtils.h"
#include "QN_MLP_BunchCudaVar.h"
#include "QN_fltvec.h"
#include "QN_intvec.h"
#include "QN_cuvec.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>


// These appear later but we do not want them in the header file
// __global__ void QN_BunchCudaVar_forward_bunch(QN_BunchCudaVar_Workspace *ws,
// 					      int n_frames);
// __global__ void QN_BunchCudaVar_train_bunch(QN_BunchCudaVar_Workspace *ws,
// 					    int n_frames);
// __device__ void QN_BunchCudaVar_forward_bunch_do(QN_BunchCudaVar_Workspace *ws,
//					      int n_frames);


QN_MLP_BunchCudaVar::QN_MLP_BunchCudaVar(int a_debug,
					 const char* a_dbgname,
					 size_t a_n_layers,
					 const size_t a_layer_units[QN_MLP_MAX_LAYERS],
					 enum QN_OutputLayerType a_outtype,
					 size_t a_size_bunch)
    : QN_MLP_BaseFl(a_debug, a_dbgname, "QN_MLP_BunchCudaVar",
		    a_size_bunch, a_n_layers,
		    a_layer_units[0], a_layer_units[1],
		    a_layer_units[2], a_layer_units[3], a_layer_units[4]),
      out_layer_type(a_outtype)
{
    size_t i;

    // Initialize CUDA if it has not happened already

    QN_cuda_init();
    
    // Some stuff so that when things go wrong it is more obvious.
    // for (i=0; i<MAX_LAYERS; i++)
    // {
    // 	layer_x[i] = NULL;
    // 	layer_y[i] = NULL;
    // 	layer_dedy[i] = NULL;
    // 	layer_dydx[i] = NULL;
    // 	layer_dedx[i] = NULL;
    // 	layer_delta_bias[i] = NULL;
    // }
    // Maybe we do not support all output layer types
    switch(out_layer_type)
    {
    case QN_OUTPUT_SOFTMAX:
    case QN_OUTPUT_SIGMOID:
    case QN_OUTPUT_SIGMOID_XENTROPY:
    case QN_OUTPUT_LINEAR:
    case QN_OUTPUT_TANH:
	break;
    default:
	clog.error("Failed to create an MLP with an invalid"
		   " output layer type.");
    }
    if (size_bunch == 0)
	clog.error("Cannot use a 0 bunch size.");


    // Allocate device data structures

    size_t in_size = layer_size[0];
    size_t out_size = layer_size[n_layers-1];

    devnew_vf("in", in_size, &(dev.in));
    devnew_vf("out", out_size, &(dev.out));
    devnew_vf("targ", out_size, &(dev.targ));

    for (i = 1; i<n_layers; i++)
    {
	size_t size = layer_size[i];
	size_t units = layer_units[i];

	devnew_vf("layer_bias", size, &(dev.layer_bias[i]));
	devnew_vf("layer_y", size, &(dev.layer_y[i]));
	devnew_vf("layer_x", size, &(dev.layer_x[i]));
	devnew_vf("layer_dedy", size, &(dev.layer_dedy[i]));
	devnew_vf("layer_dydx", size, &(dev.layer_dydx[i]));
	devnew_vf("layer_dedx", size, &(dev.layer_dedx[i]));
	devnew_vf("layer_delta_bias", units, &(dev.layer_delta_bias[i]));
    }
    // Set up the per-weight-matrix data structures.
    for (i = 0; i<n_weightmats; i++)
    {
	// Note the host weights are alloacted by QN_MLP_BaseFl 
	size_t n_weights = weights_size[i];

	// Allocate device data structures
	devnew_vf("weights", n_weights, &dev.weights[i]);
    }

    clog.log(QN_LOG_PER_RUN, "Created net with %lu layers, bunchsize %lu.",
	     n_layers, size_bunch);
    for (i=0; i<n_layers; i++)
    {
	clog.log(QN_LOG_PER_RUN, "Layer %lu has %lu units.",
		 i+1, layer_units[i]);
    }
    dev_weights_stale = QN_TRUE;
    host_weights_stale = QN_FALSE;

}

QN_MLP_BunchCudaVar::~QN_MLP_BunchCudaVar()
{
    size_t i;

    QN_cuda_check();
    // Wind down the per-weight-matrix data structures.
    for (i = 0; i<n_weightmats; i++)
    {
	// Deallocate device data structures
	devfree_vf("weights", dev.weights[i]);
	// Note the host weights are deallocated by QN_MLP_BaseFl 
    }
    // Wind down the per-layer data structures.
    for (i = 1; i<n_layers; i++)
    {
	// delete [] layer_y[i];
	// delete [] layer_delta_bias[i];
	// delete [] layer_dedx[i];
	// delete [] layer_dydx[i];
	// delete [] layer_dedy[i];
	// delete [] layer_x[i];
	// Note the host biases are deallocated by QN_MLP_BaseFl 

	devfree_vf("layer_delta_bias", dev.layer_delta_bias[i]);
	devfree_vf("layer_dedx", dev.layer_dedx[i]);
	devfree_vf("layer_dydx", dev.layer_dydx[i]);
	devfree_vf("layer_dedy", dev.layer_dedy[i]);
	devfree_vf("layer_x", dev.layer_x[i]);
	devfree_vf("layer_y", dev.layer_y[i]);
	devfree_vf("layer_bias", dev.layer_bias[i]);
    }
    devfree_vf("targ", dev.targ);
    devfree_vf("out", dev.out);
    devfree_vf("in", dev.in);
}



void
QN_MLP_BunchCudaVar::forward_bunch(size_t n_frames, const float* in, float* out)
{
//    printf("in=%x, out=%x\n", in, out);

    // Copy the data across to the device
    int in_size = n_frames * layer_units[0];
    int out_size = n_frames * layer_units[n_layers-1];
    todev_vf_vf("forward_bunch().in", in_size, in, dev.in);

    size_t cur_layer;		// The index of the current layer.
    size_t prev_layer;		// The index of the previous layer.
    size_t cur_weinum;		// The index of the current weight matrix.
    size_t cur_layer_units;	// The number of units in the current layer.
    size_t prev_layer_units;	// The number of units in the previous layer.
    size_t cur_layer_size;	// The size of the current layer.
    float* cur_layer_x;		// Input to the current layer non-linearity.
    float* cur_layer_y;		// Output from the current layer
				// non-linearity.
    const float* prev_layer_y;	// Output from the previous non-linearity.
    float* cur_layer_bias;	// Biases for the current layer.
    float* cur_weights;		// Weights inputing to the current layer.

    // Iterate over all of the layers except the input.  This is just one 
    // iteration for 2-layer MLPs.
    // Note that layer index starts at 0 for inputlayer, so we start at 1.
    for (cur_layer=1; cur_layer<n_layers; cur_layer++)
    {
	prev_layer = cur_layer - 1;
	cur_weinum = cur_layer - 1;
	cur_layer_units = layer_units[cur_layer];
	prev_layer_units = layer_units[prev_layer];
	cur_layer_size = cur_layer_units * n_frames;
	cur_layer_x = dev.layer_x[cur_layer];
	cur_layer_y = dev.layer_y[cur_layer];
	if (cur_layer==1)
	    prev_layer_y = dev.in;
	else
	    prev_layer_y = dev.layer_y[prev_layer];
	cur_layer_bias = dev.layer_bias[cur_layer];
	cur_weights = dev.weights[cur_weinum];

	if (checking)
	    devcheck("forward_bunch #1");
	qn_dev_copy_vf_mf(n_frames, cur_layer_units, cur_layer_bias,
			    cur_layer_x);
	if (checking)
	    devcheck("forward_bunch #2");
	qn_dev_mulntacc_mfmf_mf(n_frames, prev_layer_units, cur_layer_units,
				prev_layer_y, cur_weights,
				cur_layer_x); 
	if (checking)
	    devcheck("forward_bunch #3");
	
	// Check if we are doing things differently for the final layer.
	if (cur_layer!=n_layers - 1)
	{
	    // This is the intermediate layer non-linearity.
	    qn_dev_sigmoid_vf_vf(cur_layer_size, cur_layer_x,
				 cur_layer_y);
	    if (checking)
		devcheck("forward_bunch #4");
	}
	else
	{
	    // This is the output layer non-linearity.
	    switch(out_layer_type)
	    {
	    case QN_OUTPUT_SIGMOID:
	    case QN_OUTPUT_SIGMOID_XENTROPY:
		qn_dev_sigmoid_vf_vf(cur_layer_size, cur_layer_x, dev.out);
		if (checking)
		    devcheck("forward_bunch #5");
		break;
	    case QN_OUTPUT_SOFTMAX:
		qn_dev_multisoftmax_mf_mf(n_frames, cur_layer_units,
					  cur_layer_x, dev.out);
		if (checking)
		    devcheck("forward_bunch #6");
		break;
	    case QN_OUTPUT_LINEAR:
		qn_dev_copy_vf_vf(cur_layer_size, cur_layer_x, dev.out);
		if (checking)
		    devcheck("forward_bunch #7");
		break;
	    case QN_OUTPUT_TANH:
		qn_dev_tanh_vf_vf(cur_layer_size, cur_layer_x, dev.out);
		if (checking)
		    devcheck("forward_bunch #8");
		break;
	    default:
		assert(0);
	    }
	}
    }
    // Copy the data back from the device
    fromdev_vf_vf("forward_bunch().out", out_size, dev.out, out);
    if (checking)
	devcheck("forward_bunch #9");

}

void
QN_MLP_BunchCudaVar::train_bunch(size_t n_frames, const float *in,
				 const float* target, float* out)
{
// First move forward, which copies over in and out
    forward_bunch(n_frames, in, out);
    if (checking)
	devcheck("train_bunch #0");

// So we stil have to copy across targ
    int out_size = n_frames * layer_units[n_layers-1];
    todev_vf_vf("train_bunch().targ", out_size, target, dev.targ);
    if (checking)
	devcheck("train_bunch #1");

    size_t cur_layer;		// The index of the current layer.
    size_t prev_layer;		// The index of the previous layer.
    size_t cur_weinum;		// The index of the current weight matrix.
    size_t cur_layer_units;	// The number of units in the current layer.
    size_t prev_layer_units;	// The number of units in the previous layer.
    size_t cur_layer_size;	// The size of the current layer.
    float* cur_layer_y;		// Output from the current layer
				// non-linearity.
    const float* prev_layer_y;	// Output from the previous non-linearity.
    float* cur_layer_dydx;	// dydx for the current layer.
    float* cur_layer_dedy;	// dedy for the current layer.
    float* prev_layer_dedy;	// dedy for the previous layer.
    float* cur_layer_dedx;	// dedx for the current layer.
    float* cur_layer_bias;	// Biases for the current layer.
    float* cur_layer_delta_bias; // Delta biases for the current layer.
    float* cur_weights;		// Weights inputing to the current layer.


    // Iterate back over all layers but the first.
    for (cur_layer=n_layers-1; cur_layer>0; cur_layer--)
    {
	prev_layer = cur_layer - 1;
	cur_weinum = cur_layer - 1;
	cur_layer_units = layer_units[cur_layer];
	prev_layer_units = layer_units[prev_layer];
	cur_layer_size = cur_layer_units * n_frames;
	cur_layer_y = dev.layer_y[cur_layer];
	if (cur_layer==1)
	    prev_layer_y = dev.in;
	else
	    prev_layer_y = dev.layer_y[prev_layer];
	cur_layer_dydx = dev.layer_dydx[cur_layer];
	cur_layer_dedy = dev.layer_dedy[cur_layer];
	prev_layer_dedy = dev.layer_dedy[prev_layer];
	cur_layer_dedx = dev.layer_dedx[cur_layer];
	cur_layer_bias = dev.layer_bias[cur_layer];
	cur_layer_delta_bias = dev.layer_delta_bias[cur_layer];
	cur_weights = dev.weights[cur_weinum];

	float cur_neg_weight_learnrate = neg_weight_learnrate[cur_weinum];
	float cur_neg_bias_learnrate = neg_bias_learnrate[cur_layer];

	if (cur_layer!=n_layers - 1 && backprop_weights[cur_weinum+1])
	{
 	    // Propogate error back through sigmoid
 	    qn_dev_dsigmoid_vf_vf(cur_layer_size, cur_layer_y, cur_layer_dydx);
	    if (checking)
		devcheck("train_bunch #3");
 	    qn_dev_mul_vfvf_vf(cur_layer_size, cur_layer_dydx, cur_layer_dedy,
			       cur_layer_dedx);
	    if (checking)
		devcheck("train_bunch #4");
	}
	else
	{
	    // Going back through the output layer.
	    switch(out_layer_type)
	    {
	    case QN_OUTPUT_SIGMOID:
		// For a sigmoid layer, de/dx = de/dy . dy/dx
		qn_dev_sub_vfvf_vf(cur_layer_size, dev.out, dev.targ,
				   cur_layer_dedy);
		if (checking)
		    devcheck("train_bunch #5");
		qn_dev_dsigmoid_vf_vf(cur_layer_size, dev.out, cur_layer_dydx);
		if (checking)
		    devcheck("train_bunch #6");
		qn_dev_mul_vfvf_vf(cur_layer_size, cur_layer_dydx,
				   cur_layer_dedy, cur_layer_dedx);
		if (checking)
		    devcheck("train_bunch #7");
		break;
	    case QN_OUTPUT_TANH:
		// tanh output layer very similar to sigmoid
		qn_dev_sub_vfvf_vf(cur_layer_size, dev.out, dev.targ,
				   cur_layer_dedy);
		if (checking)
		    devcheck("train_bunch #8");
		qn_dev_dtanh_vf_vf(cur_layer_size, dev.out, cur_layer_dydx);
		if (checking)
		    devcheck("train_bunch #9");
		qn_dev_mul_vfvf_vf(cur_layer_size,
			       cur_layer_dydx, cur_layer_dedy, cur_layer_dedx);
		if (checking)
		    devcheck("train_bunch #10");
		break;
	    case QN_OUTPUT_SIGMOID_XENTROPY:
	    case QN_OUTPUT_SOFTMAX:
	    case QN_OUTPUT_LINEAR:
		// For these layers, dx = dy
		qn_dev_sub_vfvf_vf(cur_layer_size, dev.out, dev.targ,
				   cur_layer_dedx);
		if (checking)
		    devcheck("train_bunch #11");
		break;
	    default:
		assert(0);
	    } // End of output layer type switch.
	} // End of special output layer treatment.

	// Back propogate error through this layer.
	if (cur_layer!=1 && backprop_weights[cur_weinum])
	{
	    qn_dev_mul_mfmf_mf(n_frames, cur_layer_units, prev_layer_units,
			   cur_layer_dedx, cur_weights, prev_layer_dedy);
	    if (checking)
		devcheck("train_bunch #12");
	}
	// Update weights.
	if (cur_neg_weight_learnrate!=0.0f)
	{
	    qn_dev_multnacc_fmfmf_mf(n_frames, cur_layer_units, prev_layer_units,
				 cur_neg_weight_learnrate, cur_layer_dedx,
				 prev_layer_y, cur_weights);
	    if (checking)
		devcheck("train_bunch #13");
	}
	// Update biases.
	if (cur_neg_bias_learnrate!=0.0f)
	{
	    qn_dev_sumcol_mf_vf(n_frames, cur_layer_units, cur_layer_dedx,
				cur_layer_delta_bias); 
	    if (checking)
		devcheck("train_bunch #14");
	    qn_dev_mulacc_vff_vf(cur_layer_units, cur_layer_delta_bias,
				 cur_neg_bias_learnrate, cur_layer_bias);
	    if (checking)
		devcheck("train_bunch #15");
	}
    } // End of iteration over all layers.


    // Copy the data back from the device
    fromdev_vf_vf("train_bunch().out", out_size, dev.out, out);
    if (checking)
	devcheck("train_bunch #16");

}

void
QN_MLP_BunchCudaVar::forward(size_t n_frames, const float* in, float* out)
{
    refresh_dev_weights();
    QN_MLP_BaseFl::forward(n_frames, in, out);
}

void
QN_MLP_BunchCudaVar::train(size_t n_frames, const float* in,
			   const float* target, float* out)
{
    refresh_dev_weights();
    QN_MLP_BaseFl::train(n_frames, in, target, out);
    host_weights_stale = QN_TRUE;
}

void
QN_MLP_BunchCudaVar::set_weights(enum QN_SectionSelector which,
				 size_t row, size_t col,
				 size_t n_rows, size_t n_cols,
				 const float* weights)
{
    refresh_host_weights();
    QN_MLP_BaseFl::set_weights(which, row, col, n_rows, n_cols, weights);
    dev_weights_stale = QN_TRUE;
}


void
QN_MLP_BunchCudaVar::get_weights(enum QN_SectionSelector which,
				 size_t row, size_t col,
				 size_t n_rows, size_t n_cols,
				 float* weights)
{
    refresh_host_weights();
    QN_MLP_BaseFl::get_weights(which, row, col, n_rows, n_cols, weights);
}

void
QN_MLP_BunchCudaVar::refresh_dev_weights(void)
{
    if (dev_weights_stale)
    {
	dev_weights_stale = QN_FALSE;

	size_t i;

	for (i = 0; i<n_weightmats; i++)
	{
	    size_t n_weights;

	    n_weights = weights_size[i]; 
	    todev_vf_vf("refresh_dev_weights().weights",
			n_weights, weights[i], dev.weights[i]);
	}

	for (i = 1; i<n_layers; i++)
	{
	    size_t n_biases;

	    n_biases = layer_size[i];
	    todev_vf_vf("refresh_dev_weights().layer_bias",
			n_biases, layer_bias[i], dev.layer_bias[i]);
	}
    }
}

void
QN_MLP_BunchCudaVar::refresh_host_weights(void)
{
    if (host_weights_stale)
    {
	host_weights_stale = QN_FALSE;

	size_t i;

	for (i = 0; i<n_weightmats; i++)
	{
	    size_t n_weights;

	    n_weights = weights_size[i]; 
	    fromdev_vf_vf("refresh_host_weights.weights)",
			  n_weights, dev.weights[i], weights[i]);
	}

	for (i = 1; i<n_layers; i++)
	{
	    size_t n_biases;

	    n_biases = layer_size[i];
	    fromdev_vf_vf("freresh_host_weights().layer_bias", 
			   n_biases, dev.layer_bias[i], layer_bias[i]);
	}
    }
}

void
QN_MLP_BunchCudaVar::devnew_vf(const char* varname, int n, float **devptr)
{
    hipblasStatus_t e;

    e = cublasAlloc(n, sizeof(float), (void **) devptr);
    if (e != HIPBLAS_STATUS_SUCCESS)
    {
	clog.error("cuda blas device new_vf error variable %s - %s.",
		   varname, QN_cublas_error_string(e));
    }
    clog.log(QN_LOG_PER_EPOCH, "Created CUDA float vec \"%s\" size %i at %.8x\n", varname, n, (unsigned long) *devptr);
}

void
QN_MLP_BunchCudaVar::devnew_vi(const char* varname, int n, int **devptr)
{
    hipblasStatus_t e;

    e = cublasAlloc(n, sizeof(int), (void **) devptr);
    if (e != HIPBLAS_STATUS_SUCCESS)
    {
	clog.error("cuda blas device new_vi error variable %s - %s.",
		   varname, QN_cublas_error_string(e));
    }
    clog.log(QN_LOG_PER_EPOCH, "Created CUDA int vec \"%s\" size %i at %.8x\n", varname, n, (unsigned long) *devptr);

}


void 
QN_MLP_BunchCudaVar::devcheck(const char* location)
{
    hipError_t e;

    e = hipDeviceSynchronize();
    if (e!=hipSuccess)
    {
	clog.error("asynchronous CUDA error at %s - %s.",
		   location, hipGetErrorString(e));
    }
    
    hipblasStatus_t eb;

    eb = cublasGetError();
    if (eb!=HIPBLAS_STATUS_SUCCESS)
	QN_ERROR("QN_cuda_check", "accumulated cublas error detected");
}

void
QN_MLP_BunchCudaVar::devnew(const char* varname, int n, int size,
			    void **devptr)
{
    hipblasStatus_t e;

    e = cublasAlloc(n, size, devptr);
    if (e != HIPBLAS_STATUS_SUCCESS)
    {
	clog.error("cuda blasw device free error variable %s - %s.",
		   varname, QN_cublas_error_string(e));
    }

}

void
QN_MLP_BunchCudaVar::devfree(const char* varname, const void* devptr)
{
    hipblasStatus_t e;
    e = cublasFree(devptr);
    if (e != HIPBLAS_STATUS_SUCCESS)
    {
	clog.error("cuda blas device free error variable %s - %s.",
		   varname, QN_cublas_error_string(e)); 
    }
}

void
QN_MLP_BunchCudaVar::devfree_vf(const char* varname, const float* devptr)
{
    hipblasStatus_t e;
    e = cublasFree((void *) devptr);
    if (e != HIPBLAS_STATUS_SUCCESS)
    {
	clog.error("cuda blas device free_vf error variable %s - %s.",
		   varname, QN_cublas_error_string(e)); 
    }
}

void
QN_MLP_BunchCudaVar::devfree_vi(const char* varname, const int* devptr)
{
    hipblasStatus_t e;
    e = cublasFree((void *) devptr);
    if (e != HIPBLAS_STATUS_SUCCESS)
    {
	clog.error("cuda blas device free_vf error variable %s - %s.",
		   varname, QN_cublas_error_string(e)); 
    }
}

void
QN_MLP_BunchCudaVar::todev_vf_vf(const char* varname, int n, const float* from,
				 float* devto)
{
    hipblasStatus_t e;

    e = hipblasSetVector(n, sizeof(float), from, 1, devto, 1);
    if (e != HIPBLAS_STATUS_SUCCESS)
    {
	clog.error("cuda blas todev_vf_vf error variable %s - %s.",
		   varname, QN_cublas_error_string(e)); 
    }
    clog.log(QN_LOG_PER_BUNCH, "Copied %i floats to device variable \"%s\" at address %.8x\n", n, varname, devto);
}

void
QN_MLP_BunchCudaVar::fromdev_vf_vf(const char* varname, int n,
				   const float* devfrom, float* to)
{
    hipblasStatus_t e;

    e = hipblasGetVector(n, sizeof(float), devfrom, 1, to, 1);
    if (e != HIPBLAS_STATUS_SUCCESS)
    {
	clog.error("cuda blas fromdev_vf_vf error variable %s - %s.",
		   varname, QN_cublas_error_string(e)); 
    }
    clog.log(QN_LOG_PER_BUNCH, "Copied %i floats from device variable \"%s\" at address %.8x\n", n, varname, devfrom);
}

void
QN_MLP_BunchCudaVar::todev_vi_vi(const char* varname, int n,
				 const int* from, int* devto)
{
    hipblasStatus_t e;

    e = hipblasSetVector(n, sizeof(int), from, 1, devto, 1);
    if (e != HIPBLAS_STATUS_SUCCESS)
    {
	clog.error("cuda blas todev_vi_vi error variable %s - %s.",
		   varname, QN_cublas_error_string(e)); 
    }
    clog.log(QN_LOG_PER_BUNCH, "Copied %i ints to device variable \"%s\" at address %.8x\n", n, varname, devto);
}

void
QN_MLP_BunchCudaVar::fromdev_vi_vi(const char* varname, int n,
				   const int* devfrom, int* to)
{
    hipblasStatus_t e;

    e = hipblasGetVector(n, sizeof(int), devfrom, 1, to, 1);
    if (e != HIPBLAS_STATUS_SUCCESS)
    {
	clog.error("cuda blas fromdev_vi_vi error variable %s - %s.",
		   varname, QN_cublas_error_string(e)); 
    }
    clog.log(QN_LOG_PER_BUNCH, "Copied %i ints from device variable \"%s\" at address %.8x\n", n, varname, devfrom);
}




